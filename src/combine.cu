#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>
#include <sstream>
#include <fstream>

#define MAX_DIMS 10
#define TILE 32
#define BASE_THREAD_NUM 32
#define BLOCK_DIM 1024
typedef float scalar_t;

#define ADD_FUNC       1
#define MUL_FUNC       2
#define ID_FUNC        3
#define NEG_FUNC       4
#define LT_FUNC        5
#define EQ_FUNC        6
#define SIGMOID_FUNC   7
#define RELU_FUNC      8
#define RELU_BACK_FUNC 9
#define LOG_FUNC       10
#define LOG_BACK_FUNC  11
#define EXP_FUNC       12
#define INV_FUNC       13
#define INV_BACK_FUNC  14
#define IS_CLOSE_FUNC  15
#define MAX_FUNC       16
#define POW            17
#define TANH           18

__device__ float fn(int fn_id, float x, float y=0) {
    switch(fn_id) {
      case ADD_FUNC: {
        return x + y;
      }
      case MUL_FUNC: {
        return x * y;
      }
      case ID_FUNC: {
      	return x;
      }
      case NEG_FUNC: {
        return -x;
      }
      case LT_FUNC: {
        if (x < y) {
          return 1.0;
        }
        else {
          return 0.0;
        }
      }
      case EQ_FUNC: {
        if (x == y) {
          return 1.0;
        }
        else {
          return 0.0;
        }
      }
      case SIGMOID_FUNC: {
        if (x >= 0) {
          return 1.0 / (1.0 + exp(-x));
        }
        else {
          return exp(x) / (1.0 + exp(x));
        }
      }
      case RELU_FUNC: {
        return max(x, 0.0);
      }
      case RELU_BACK_FUNC: {
        if (x > 0) {
          return y;
        }
        else {
          return 0.0;
        }
      }
      case LOG_FUNC: {
        return log(x + 1e-6);
      }
      case LOG_BACK_FUNC: {
        return y / (x + 1e-6);
      }
      case EXP_FUNC: {
        return exp(x);
      }
      case INV_FUNC: {
        return float(1.0 / x);
      }
      case INV_BACK_FUNC: {
        return -(1.0 / (x * x)) * y;
      }
      case IS_CLOSE_FUNC: {
        return (x - y < 1e-2) && (y - x < 1e-2);
      }
      case MAX_FUNC: {
        if (x > y) {
          return x;
        }
        else {
          return y;
        }
      }
      case POW: {
	// TODO
        return powf(x, y);
      }
      case TANH: {
	// TODO
        return tanhf(x);
      }
      default: {
        return x + y;
      }
    }
    
}


__device__ int index_to_position(const int* index, const int* strides, int num_dims) {
    int position = 0;
    for (int i = 0; i < num_dims; ++i) {
        position += index[i] * strides[i];
    }
    return position;
}

__device__ void to_index(int ordinal, const int* shape, int* out_index, int num_dims) {
    int cur_ord = ordinal;
    for (int i = num_dims - 1; i >= 0; --i) {
        int sh = shape[i];
        out_index[i] = cur_ord % sh;
        cur_ord /= sh;
    }
}

__device__ void broadcast_index(const int* big_index, const int* big_shape, int num_dims_big, int* out_index, const int* shape, int num_dims) {
  /**
   * Convert a big_index into big_shape to a smaller out_index into shape following broadcasting rules.
   * In this case it may be larger or with more dimensions than the shape given.
   * Additional dimensions may need to be mapped to 0 or removed.
   *
   * Args:
   *    big_index: multidimensional index of bigger tensor
   *    big_shape: tensor shape of bigger tensor
   *    nums_big_dims: number of dimensions in bigger tensor
   *    out_index: multidimensional index of smaller tensor
   *    shape: tensor shape of smaller tensor
   *    num_dims: number of dimensions in smaller tensor
   *
   * Returns:
   *    None (Fills in out_index)
  */
    for (int i = 0; i < num_dims; ++i) {
        if (shape[i] > 1) {
            out_index[i] = big_index[i + (num_dims_big - num_dims)];
        } else {
            out_index[i] = 0;
        }
    }
}
__global__ void SGMVKernel(
  scalar_t* out,
  const int* out_shape,
  const int* out_strides,
  scalar_t* a_storage,
  const int* a_shape,
  const int* a_strides,
  scalar_t* b_storage,
  const int* b_shape,
  const int* b_strides,
  const int* lora_idx_s
) {
  // general SGMV kernal, call twice with different a and b can work as expand or shrink kernal
  int n_lora = blockDim.z;
  int batch_size_offset = lora_idx_s[blockIdx.z];
  int row_limit[2]; //dim order: (row, col). row_limit = {lower_bound_inclusive, upper_bound_exclusive}
  // this is the row idx limit of the output matrix
  row_limit[0] = lora_idx_s[blockIdx.z]; // inclusive. lora_idx_s[blockIdx.z] is the start of current group
  row_limit[1] = lora_idx_s[blockIdx.z+1]; // exclusive



  ////////////////// mat mult: a*b /////////////////////


  __shared__ scalar_t a_shared[TILE][TILE];
  __shared__ scalar_t b_shared[TILE][TILE];

  // In each block, we will compute a batch of the output matrix
  // All the threads in the block will work together to compute this batch
  int batch = blockIdx.z;



  /// BEGIN ASSIGN1_2
  /// TODO
  // Hints:
  int temp_pos_2d[2];
  int temp_pos_3d[3];
  //int int_index[MAX_DIMS];

  // 1. Compute the row and column of the output matrix this block will compute
  int i = batch_size_offset + blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  // 2. Compute the position in the output array that this thread will write to

  int N = a_shape[2]; //N here means m, the second dim of input x

  // 3. Iterate over tiles of the two input matrices, read the data into shared memory
  int tile_i = threadIdx.x;
  int tile_j = threadIdx.y;
  double out_temp = 0;
  for(int ks = 0; ks < N; ks+=TILE){
      temp_pos_2d[0] = i;
      temp_pos_2d[1] = ks+tile_j;
      if(i<row_limit[1] && otemp_pos_2d[1] <N){
          a_shared[tile_i][tile_j] = a_storage[index_to_position(temp_pos_2d, a_strides, 2)];
      }else
          a_shared[tile_i][tile_j] = 0;
      temp_pos_3d[0] = blockIdx.z;
      temp_pos_3d[1] = ks+tile_i;
      temp_pos_3d[2] = j;
      if (temp_pos_3d[1] < N && j < b_shape[2]){
          b_shared[tile_i][tile_j] = b_storage[index_to_position(temp_pos_3d, b_strides, 3)];
      }else
          b_shared[tile_i][tile_j] = 0;

      __syncthreads();
      for(int ki = 0; ki < TILE; ki++){
          out_temp += a_shared[tile_i][ki] * b_shared[ki][tile_j];
      }

      __syncthreads();
  }
  if (i<row_limit[1] && j<out_shape[1]){
      temp_pos_2d[0] = i;
      temp_pos_2d[1] = j;
      out[index_to_position(temp_pos_2d, out_strides, 2)] = out_temp;

  }
}

__global__ void MatrixMultiplyKernel(
    scalar_t* out,
    const int* out_shape,
    const int* out_strides,
    scalar_t* a_storage,
    const int* a_shape,
    const int* a_strides,
    scalar_t* b_storage,
    const int* b_shape,
    const int* b_strides
) {
  /**
   * Multiply two (compact) matrices into an output (also comapct) matrix. Matrix a and b are both in a batch
   * format, with shape [batch_size, m, n], [batch_size, n, p].
   * Requirements:
   * - All data must be first moved to shared memory.
   * - Only read each cell in a and b once.
   * - Only write to global memory once per kernel.
   * There is guarantee that a_shape[0] == b_shape[0], a_shape[2] == b_shape[1],
   * and out_shape[0] == a_shape[0], out_shape[1] == b_shape[1]
   *
   * Args:
   *   out: compact 1D array of size batch_size x m x p to write the output to
   *   out_shape: shape of the output array
   *   out_strides: strides of the output array
   *   a_storage: compact 1D array of size batch_size x m x n
   *   a_shape: shape of the a array
   *   a_strides: strides of the a array
   *   b_storage: comapct 2D array of size batch_size x n x p
   *   b_shape: shape of the b array
   *   b_strides: strides of the b array
   *
   * Returns:
   *   None (Fills in out array)
   */


    __shared__ scalar_t a_shared[TILE][TILE];
    __shared__ scalar_t b_shared[TILE][TILE];

    // In each block, we will compute a batch of the output matrix
    // All the threads in the block will work together to compute this batch
    int batch = blockIdx.z;
    int a_batch_stride = a_shape[0] > 1 ? a_strides[0] : 0;
    int b_batch_stride = b_shape[0] > 1 ? b_strides[0] : 0;


    /// BEGIN ASSIGN1_2
    /// TODO
    // Hints:
    int out_index[MAX_DIMS];
    //int int_index[MAX_DIMS];
    int out_size = 1;
    for(int i=0; i<3; ++i){
        if(out_shape[i]!=0)
            out_size*=out_shape[i];
    }
    // 1. Compute the row and column of the output matrix this block will compute
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    // 2. Compute the position in the output array that this thread will write to
    out_index[0] = batch;
    out_index[1] = i;
    out_index[2] = j;

    int N = a_shape[2];

    // 3. Iterate over tiles of the two input matrices, read the data into shared memory
    int tile_i = threadIdx.x;
    int tile_j = threadIdx.y;
    double out_temp = 0;
    for(int ks = 0; ks < N; ks+=TILE){
        out_index[0] = batch;
        out_index[1] = i;
        out_index[2] = ks+tile_j;
        if(i<a_shape[1] && out_index[2] <N){
            a_shared[tile_i][tile_j] = a_storage[index_to_position(out_index, a_strides, 3)];
        }else
            a_shared[tile_i][tile_j] = 0;
        out_index[1] = ks+tile_i;
        out_index[2] = j;
        if (out_index[1] < N && j < b_shape[2]){
            b_shared[tile_i][tile_j] = b_storage[index_to_position(out_index, b_strides, 3)];
        }else
            b_shared[tile_i][tile_j] = 0;

        __syncthreads();
        for(int ki = 0; ki < TILE; ki++){

            if(i==32 && j==0 && tile_i ==0&&ki==0)
                // printf("a: %f , b: %f \n", a_shared[tile_i][ki], b_shared[ki][tile_j]);
            out_temp += a_shared[tile_i][ki] * b_shared[ki][tile_j];
        }

        __syncthreads();
        if(i==32 && j==0)
          // printf("%f \n", out_temp);
    }
    if (i<out_shape[1] && j<out_shape[2]){
        out_index[1] = i;
        out_index[2] = j;
        out[index_to_position(out_index, out_strides, 3)] = out_temp;//out_temp;

    }

    /// END ASSIGN1_2
}


__global__ void mapKernel(
    scalar_t* out,
    int* out_shape,
    int* out_strides,
    int out_size,
    scalar_t* in_storage,
    int* in_shape,
    int* in_strides,
    int shape_size,
    int fn_id
) {
  /**
   * Map function. Apply a unary function to each element of the input array and store the result in the output array.
   * Optimization: Parallelize over the elements of the output array.
   *
   * You may find the following functions useful:
   * - index_to_position: converts an index to a position in a compact array
   * - to_index: converts a position to an index in a multidimensional array
   * - broadcast_index: converts an index in a smaller array to an index in a larger array
   *
   * Args:
   *  out: compact 1D array of size out_size to write the output to
   *  out_shape: shape of the output array
   *  out_strides: strides of the output array
   *  out_size: size of the output array
   *  in_storage: compact 1D array of size in_size
   *  in_shape: shape of the input array
   *  in_strides: strides of the input array
   *  shape_size: number of dimensions in the input and output arrays, assume dimensions are the same
   *  fn_id: id of the function to apply to each element of the input array
   *
   * Returns:
   *  None (Fills in out array)
   */
  int out_index[MAX_DIMS];
  int in_index[MAX_DIMS];

  /// BEGIN ASSIGN1_2
  /// TODO
  // Hints:
  // 1. Compute the position in the output array that this thread will write to
  int pos = blockIdx.x * blockDim.x + threadIdx.x;
  if(pos>=out_size) return;
  // 2. Convert the position to the out_index according to out_shape
  to_index(pos, out_shape, out_index, shape_size);
  // 3. Broadcast the out_index to the in_index according to in_shape (optional in some cases)
  broadcast_index(out_index, out_shape, shape_size, in_index, in_shape, shape_size);
  //  broadcast_index(const int* big_index, const int* big_shape, int num_dims_big, int* out_index, const int* shape, int num_dims)
  // 4. Calculate the position of element in in_array according to in_index and in_strides
  int in_pos = index_to_position(in_index, in_strides, shape_size);
  // 5. Calculate the position of element in out_array according to out_index and out_strides

  // 6. Apply the unary function to the input element and write the output to the out memory
  out[pos] = fn(fn_id, in_storage[in_pos]);
  // assert(false && "Not Implemented");
  /// END ASSIGN1_2
  /// END ASSIGN1_2
}


__global__ void reduceKernel(
    scalar_t* out,
    int* out_shape,
    int* out_strides,
    int out_size,
    scalar_t* a_storage,
    int* a_shape,
    int* a_strides,
    int reduce_dim,
    double reduce_value,
    int shape_size,
    int fn_id
) {
  /**
   * Reduce function. Apply a reduce function to elements of the input array a and store the result in the output array.
   * Optimization:
   * Parallelize over the reduction operation. Each kernel performs one reduction.
   * e.g. a = [[1, 2, 3], [4, 5, 6]], kernel0 computes reduce([1, 2, 3]), kernel1 computes reduce([4, 5, 6]).
   *
   * You may find the following functions useful:
   * - index_to_position: converts an index to a position in a compact array
   * - to_index: converts a position to an index in a multidimensional array
   *
   * Args:
   *  out: compact 1D array of size out_size to write the output to
   *  out_shape: shape of the output array
   *  out_strides: strides of the output array
   *  out_size: size of the output array
   *  a_storage: compact 1D array of size in_size
   *  a_shape: shape of the input array
   *  a_strides: strides of the input array
   *  reduce_dim: dimension to reduce on
   *  reduce_value: initial value for the reduction
   *  shape_size: number of dimensions in the input & output array, assert dimensions are the same
   *  fn_id: id of the reduce function, currently only support add, multiply, and max
   *
   *
   * Returns:
   *  None (Fills in out array)
   */

    // __shared__ double cache[BLOCK_DIM]; // Uncomment this line if you want to use shared memory to store partial results
    int out_index[MAX_DIMS];

    /// BEGIN ASSIGN1_2
    /// TODO
    // 1. Define the position of the output element that this thread or this block will write to
    int pos = blockIdx.x * blockDim.x + threadIdx.x;
    // 2. Convert the out_pos to the out_index according to out_shape
    if(pos>=out_size) return;
    to_index(pos, out_shape, out_index, shape_size);
    // 3. Initialize the reduce_value to the output element
    out[pos] = reduce_value;
    // 4. Iterate over the reduce_dim dimension of the input array to compute the reduced value
    for(int i=0; i<a_shape[reduce_dim]; ++i){
        out_index[reduce_dim] = i;
        int a_pos = index_to_position(out_index, a_strides, shape_size);
        out[pos] = fn(fn_id, out[pos], a_storage[a_pos]);
    }
    /// END ASSIGN1_2
}

__global__ void zipKernel(
    scalar_t* out,
    int* out_shape,
    int* out_strides,
    int out_size,
    int out_shape_size,
    scalar_t* a_storage,
    int* a_shape,
    int* a_strides,
    int a_shape_size,
    scalar_t* b_storage,
    int* b_shape,
    int* b_strides,
    int b_shape_size,
    int fn_id
) {
  /**
   * Zip function. Apply a binary function to elements of the input array a & b and store the result in the output array.
   * Optimization: Parallelize over the elements of the output array.
   *
   * You may find the following functions useful:
   * - index_to_position: converts an index to a position in a compact array
   * - to_index: converts a position to an index in a multidimensional array
   * - broadcast_index: converts an index in a smaller array to an index in a larger array
   *
   * Args:
   *  out: compact 1D array of size out_size to write the output to
   *  out_shape: shape of the output array
   *  out_strides: strides of the output array
   *  out_size: size of the output array
   *  out_shape_size: number of dimensions in the output array
   *  a_storage: compact 1D array of size in_size
   *  a_shape: shape of the input array
   *  a_strides: strides of the input array
   *  a_shape_size: number of dimensions in the input array
   *  b_storage: compact 1D array of size in_size
   *  b_shape: shape of the input array
   *  b_strides: strides of the input array
   *  b_shape_size: number of dimensions in the input array
   *  fn_id: id of the function to apply to each element of the a & b array
   *
   *
   * Returns:
   *  None (Fills in out array)
   */

    int out_index[MAX_DIMS];
    int a_index[MAX_DIMS];
    int b_index[MAX_DIMS];

    /// BEGIN ASSIGN1_2
    /// TODO
    // Hints:
    int pos = blockIdx.x * blockDim.x + threadIdx.x;
    if(pos>=out_size) return;
    // 2. Convert the position to the out_index according to out_shape
    to_index(pos, out_shape, out_index, out_shape_size);
    // 3. Calculate the position of element in out_array according to out_index and out_strides
    // 4. Broadcast the out_index to the a_index according to a_shape
    broadcast_index(out_index, out_shape, out_shape_size, a_index, a_shape, a_shape_size);
    // 5. Calculate the position of element in a_array according to a_index and a_strides
    int a_pos = index_to_position(a_index, a_strides, a_shape_size);
    // 6. Broadcast the out_index to the b_index according to b_shape
    broadcast_index(out_index, out_shape, out_shape_size, b_index, b_shape, b_shape_size);
    // 7.Calculate the position of element in b_array according to b_index and b_strides
    int b_pos = index_to_position(b_index, b_strides, b_shape_size);
    // 8. Apply the binary function to the input elements in a_array & b_array and write the output to the out memory
    out[pos] = fn(fn_id, a_storage[a_pos], b_storage[b_pos]); // , b_storage[b_pos]

    /// END ASSIGN1_2
}

/*

__global__ void MatrixMultiplyKernel(
    float* out,
    const int* out_shape,
    const int* out_strides,
    float* a_storage,
    const int* a_shape,
    const int* a_strides,
    float* b_storage,
    const int* b_shape,
    const int* b_strides
) {

    assert(false && "Not Implemented");
}


__global__ void mapKernel(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* in_storage, 
    int* in_shape, 
    int* in_strides,
    int shape_size,
    int fn_id
) {
    assert(false && "Not Implemented");
}


__global__ void reduceKernel(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* a_storage, 
    int* a_shape, 
    int* a_strides, 
    int reduce_dim,
    float reduce_value,
    int shape_size,
    int fn_id
) {
    assert(false && "Not Implemented");
}

__global__ void zipKernel(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size,
    int out_shape_size,
    float* a_storage, 
    int* a_shape, 
    int* a_strides,
    int a_shape_size,
    float* b_storage, 
    int* b_shape, 
    int* b_strides,
    int b_shape_size,
    int fn_id
) {
    assert(false && "Not Implemented");
}
 */


extern "C" {
  int maxLoraGroupSize(int* lora_idx_s, int length) {
    if (lora_idx_s == NULL || length <= 1) {
        // No max distance can be calculated with fewer than 2 elements
        printf("## error: lora_idx_s <= 1")
        return -1;
    }

    int maxDist = 0;

    for (int i = 1; i < length; i++) {
        // Calculate the distance between the current and previous index
        int currentDist = lora_idx_s[i] - lora_idx_s[i - 1];

        // Update maxDist if the current distance is larger
        if (currentDist > maxDist) {
            maxDist = currentDist;
        }
    }

    return maxDist;
}
  void launchSGMV(
    float* in_storage,
    int* in_shape,
    int* in_strides,
    float* out,
    int* out_shape,
    int* out_strides,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    float* b_storage,
    int* b_shape,
    int* b_strides,
    int* lora_idx_s,
    int m, int p 
) {
    // in matrix should be batch_size * m
    // a and b (lora matrixs) should be n_lora * m * n and n_lora * n * p
    // m and p here means the in_dim and out_dim of actural linear weight matrix
    // lora_idx_s: array of idxs, 0<=lora_idx_s[i]<batch_size, represent the start idx of each lora input group.
    // e.g., input 0~7: lora A, input 8~10: lora B. lora_idx_s: [0, 7, 10]
    // m: input hidden_dim
    int batch = in_shape[0];
    int n_lora = a_shape[0];
    int n = a_shape[2];
    // max_lora_group_size: the size (number of tokens) of the largest lora group in input matrix 
    int max_lora_group_size = maxLoraGroupSize(lora_idx_s, n_lora+1);
    // n means lora rank (low rank space)
    // Allocate device memory
    float *d_out, *d_a, *d_v, *d_b, *d_in;
    hipMalloc(&d_in, batch * m * sizeof(float));
    hipMalloc(&d_a, n_lora * m * n * sizeof(float));
    hipMalloc(&d_b, n_lora * n * p * sizeof(float));
    hipMalloc(&d_v, batch * n * sizeof(float));
    hipMalloc(&d_out, batch * p * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides, *d_b_shape, *d_b_strides, *d_in_shape, *d_in_strides, *d_lora_idx_s;
    hipMalloc(&d_in_shape, 2 * sizeof(int));
    hipMalloc(&d_in_strides, 2 * sizeof(int));
    hipMalloc(&d_out_shape, 2 * sizeof(int));
    hipMalloc(&d_out_strides, 2 * sizeof(int));
    hipMalloc(&d_a_shape, 3 * sizeof(int));
    hipMalloc(&d_a_strides, 3 * sizeof(int));
    hipMalloc(&d_b_shape, 3 * sizeof(int)); // leave a, b as 3d because first dim is n_lora
    hipMalloc(&d_b_strides, 3 * sizeof(int));
    hipMalloc(&d_lora_idx_s, (n_lora+1) * sizeof(int)); //+1 because the first element is 0
    hipMalloc(&d_v_shape, 2 * sizeof(int));
    hipMalloc(&d_v_strides, 2 * sizeof(int));
    int v_shape[2] = {batch, n};
    int v_strides[2] = {n, 1};
    // Copy data to the device
    hipMemcpy(d_in, in_storage, batch * m * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_a, a_storage, n_lora * m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_storage, n_lora * n * p * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_in_shape, in_shape, 2 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_strides, in_strides, 2 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, 2 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, 2 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_shape, b_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_lora_idx_s, lora_idx_s, (n_lora+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_v_shape, &v_shape, 2 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_v_strides, &v_strides, 2 * sizeof(int), hipMemcpyHostToDevice);
    
    int threadsPerBlock = BASE_THREAD_NUM;
    dim3 blockDims(threadsPerBlock, threadsPerBlock, 1); // Adjust these values based on your specific requirements
    dim3 gridDims((max_lora_group_size + threadsPerBlock - 1) / threadsPerBlock, (n + threadsPerBlock - 1) / threadsPerBlock, n_lora);
    SGMVKernel<<<gridDims, blockDims>>>(
        d_v, d_v_shape, d_v_strides, d_in, d_in_shape, d_in_strides, d_a, d_a_shape, d_a_strides, d_lora_idx_s
    );
    hipDeviceSynchronize();
    dim3 gridDims2((max_lora_group_size + threadsPerBlock - 1) / threadsPerBlock, (p + threadsPerBlock - 1) / threadsPerBlock, n_lora);
    
    SGMVKernel<<<gridDims2, blockDims>>>(
        d_out, d_out_shape, d_out_strides, d_v, d_v_shape, d_v_strides, d_b, d_b_shape, d_b_strides, d_lora_idx_s
    );

    // Copy back to the host
    hipMemcpy(out, d_out, batch * m * p * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Matmul Error: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
    hipFree(d_b_shape);
    hipFree(d_b_strides);
    hipFree(d_in);
    hipFree(d_v);
    hipFree(d_in_shape);
    hipFree(d_in_strides);
    hipFree(d_lora_idx_s);
    hipFree(d_v_shape);
    hipFree(d_v_strides);
}

void MatrixMultiply(
    float* out,
    int* out_shape,
    int* out_strides,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    float* b_storage,
    int* b_shape,
    int* b_strides,
    int batch, int m, int p
) {
    int n = a_shape[2];

    // Allocate device memory
    float *d_out, *d_a, *d_b;
    hipMalloc(&d_a, batch * m * n * sizeof(float));
    hipMalloc(&d_b, batch * n * p * sizeof(float));
    hipMalloc(&d_out, batch * m * p * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides, *d_b_shape, *d_b_strides;
    hipMalloc(&d_out_shape, 3 * sizeof(int));
    hipMalloc(&d_out_strides, 3 * sizeof(int));
    hipMalloc(&d_a_shape, 3 * sizeof(int));
    hipMalloc(&d_a_strides, 3 * sizeof(int));
    hipMalloc(&d_b_shape, 3 * sizeof(int));
    hipMalloc(&d_b_strides, 3 * sizeof(int));


    // Copy data to the device
    hipMemcpy(d_a, a_storage, batch * m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_storage, batch * n * p * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_shape, b_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, 3 * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = BASE_THREAD_NUM;
    dim3 blockDims(threadsPerBlock, threadsPerBlock, 1); // Adjust these values based on your specific requirements
    dim3 gridDims((m + threadsPerBlock - 1) / threadsPerBlock, (p + threadsPerBlock - 1) / threadsPerBlock, batch);
    MatrixMultiplyKernel<<<gridDims, blockDims>>>(
        d_out, d_out_shape, d_out_strides, d_a, d_a_shape, d_a_strides, d_b, d_b_shape, d_b_strides
    );

    // Copy back to the host
    hipMemcpy(out, d_out, batch * m * p * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Matmul Error: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
    hipFree(d_b_shape);
    hipFree(d_b_strides);
}

void tensorMap(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* in_storage, 
    int* in_shape, 
    int* in_strides,
    int in_size,
    int shape_size,
    int fn_id
) {

    float *d_out, *d_in;
    hipMalloc(&d_out, out_size * sizeof(float));
    hipMalloc(&d_in, in_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_in_shape, *d_in_strides;
    hipMalloc(&d_out_shape, shape_size * sizeof(int));
    hipMalloc(&d_out_strides, shape_size * sizeof(int));
    hipMalloc(&d_in_shape, shape_size * sizeof(int));
    hipMalloc(&d_in_strides, shape_size * sizeof(int));

    hipMemcpy(d_in, in_storage, in_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_shape, in_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_strides, in_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    
    int threadsPerBlock = BASE_THREAD_NUM;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    mapKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_out, d_out_shape, d_out_strides, out_size, 
      d_in, d_in_shape, d_in_strides, 
      shape_size, fn_id);
    
    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Map Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_in_shape);
    hipFree(d_in_strides);
}


void tensorZip(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size,
    int out_shape_size,
    float* a_storage, 
    int* a_shape, 
    int* a_strides,
    int a_size,
    int a_shape_size,
    float* b_storage, 
    int* b_shape, 
    int* b_strides,
    int b_size,
    int b_shape_size,
    int fn_id
) {

    // Allocate device memory
    float *d_out, *d_a, *d_b;
    hipMalloc((void **)&d_a, a_size * sizeof(float));
    hipMalloc(&d_b, b_size * sizeof(float));
    hipMalloc(&d_out, out_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides, *d_b_shape, *d_b_strides;
    hipMalloc(&d_out_shape, out_shape_size * sizeof(int));
    hipMalloc(&d_out_strides, out_shape_size * sizeof(int));
    hipMalloc(&d_a_shape, a_shape_size * sizeof(int));
    hipMalloc(&d_a_strides, a_shape_size * sizeof(int));
    hipMalloc(&d_b_shape, b_shape_size * sizeof(int));
    hipMalloc(&d_b_strides, b_shape_size * sizeof(int));

    // Copy data to the device
    hipMemcpy(d_a, a_storage, a_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_storage, b_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, out_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, out_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, a_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, a_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_shape, b_shape, b_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, b_shape_size * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = BASE_THREAD_NUM;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    zipKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_out, d_out_shape, d_out_strides, out_size, out_shape_size,
      d_a, d_a_shape, d_a_strides, a_shape_size,
      d_b, d_b_shape, d_b_strides, b_shape_size,
      fn_id);

    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();


    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Zip Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
    hipFree(d_b_shape);
    hipFree(d_b_strides);
}



void tensorReduce(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* a_storage, 
    int* a_shape, 
    int* a_strides, 
    int reduce_dim, 
    float reduce_value,
    int shape_size,
    int fn_id
) {
    int a_size = out_size * a_shape[reduce_dim];
    float *d_out, *d_a;
    hipMalloc(&d_out, out_size * sizeof(float));
    hipMalloc(&d_a, a_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides;
    hipMalloc(&d_out_shape, shape_size * sizeof(int));
    hipMalloc(&d_out_strides, shape_size * sizeof(int));
    hipMalloc(&d_a_shape, shape_size * sizeof(int));
    hipMalloc(&d_a_strides, shape_size * sizeof(int));

    hipMemcpy(d_a, a_storage, a_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    
    int threadsPerBlock = BASE_THREAD_NUM;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    reduceKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_out, d_out_shape, d_out_strides, out_size, 
        d_a, d_a_shape, d_a_strides, 
        reduce_dim, reduce_value, shape_size, fn_id
    );

    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Reduce Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    hipFree(d_a);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
}

}